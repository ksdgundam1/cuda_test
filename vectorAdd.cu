#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <ctime>

#define SIZE 5

__global__ void addGPU(int* d, const int* a, const int* b, const int* c)
{
    int i = threadIdx.x;
    d[i] = a[i] + b[i] + c[i];
}

int main()
{
    srand((unsigned int)time(NULL));

    int a[SIZE], b[SIZE], c[SIZE];

    for (int i = 0; i < SIZE; i++)
    {
        a[i] = rand() % 10;
        b[i] = rand() % 100;
        c[i] = rand() % 1000;
    }

    int d[SIZE] = { 0 };

    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    int* dev_d = 0;

    hipMalloc((void**)&dev_a, SIZE * sizeof(int));
    hipMalloc((void**)&dev_b, SIZE * sizeof(int));
    hipMalloc((void**)&dev_c, SIZE * sizeof(int));
    hipMalloc((void**)&dev_d, SIZE * sizeof(int));

    hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, SIZE * sizeof(int), hipMemcpyHostToDevice);

    addGPU << <1, SIZE >> > (dev_d, dev_a, dev_b, dev_c);
    //hipDeviceSynchronize();

    hipMemcpy(d, dev_d, SIZE * sizeof(int), hipMemcpyDeviceToHost);

    printf("   {%d, %d, %d, %d, %d}\n + {%d, %d, %d, %d, %d}\n + {%d, %d, %d, %d, %d}\n = {%d, %d, %d, %d, %d}\n", a[0], a[1], a[2], a[3], a[4],
        b[0], b[1], b[2], b[3], b[4],
        c[0], c[1], c[2], c[3], c[4],
        d[0], d[1], d[2], d[3], d[4]);

    hipFree(dev_a);      //Device���� �����Ҵ���� �޸� ��ȯ
    hipFree(dev_b);
    hipFree(dev_c);
    hipFree(dev_d);

    return 0;
}